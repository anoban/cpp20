#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <numeric>
#include <ranges>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>

int main(void) {
    srand(time(nullptr));

    [[maybe_unused]] int32_t dummy {};

    auto                     host_vec { thrust::host_vector<int32_t> {} };
    host_vec.reserve(10'000 * sizeof(int32_t));
    for (const int32_t& _ : std::ranges::views::iota(0, 9'000)) host_vec.push_back(rand());

    thrust::device_vector<decltype(dummy)> dev_vec { host_vec };
    auto                                   dev_sum { thrust::reduce(dev_vec.cbegin(), dev_vec.cend(), 0, thrust::plus<int32_t> {}) };
    auto                                   host_sum { thrust::reduce(host_vec.cbegin(), host_vec.cend()) };
    ::wprintf_s(L"Sum of host vector is %10d and Sum of device vector is %10d\n", host_sum, dev_sum);

    return EXIT_SUCCESS;
}