#include "hip/hip_runtime.h"
#include <concepts>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

namespace cuda {
    template<typename T> requires std::integral<T> || std::floating_point<T>
    __global__ void sum(_In_ const std::vector<T>& vector, _Inout_ T& sum) {
        sum = std::accumulate<T>(vector.cbegin(), vector.cend(), static_cast<T>(0), std::plus);
        return;
    }
} // namespace cuda

static constexpr std::size_t NUMELEMS { 100'000 };

int                          main(void) {
    auto rgenerator { std::random_device {} }; // non-deterministic random number generator
    auto mersenne { std::mt19937_64 { rgenerator() } }; // seeding the mersenne twister, use a seed for deterministic behaviour
    std::vector<int64_t> randoms(NUMELEMS * sizeof(int64_t));
    std::transform(randoms.begin(), randoms.end(), randoms.begin(), mersenne);
}