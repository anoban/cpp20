#include "hip/hip_runtime.h"
#include <algorithm>
#include <concepts>
#include <cstdlib>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

template<typename T> requires std::integral<T> || std::floating_point<T> __global__ void SumKernel(
    _In_ const thrust::device_vector<T>& in_vector_0,
    _In_ const thrust::device_vector<T>& in_vector_1,
    _Inout_ thrust::device_vector<T>& out_vector
) {
    const auto tId { threadIdx.x };
    out_vector[tId] = in_vector_0[tId] + in_vector_1[tId];
    return;
}

template<typename T> requires std::is_integral<T>::value || std::is_floating_point<T>::value
static inline thrust::host_vector<T> KernelManager(
    _In_ const std::vector<T>& in_vector_0, _In_ const std::vector<T>& in_vector_1, _Inout_ const std::vector<T>& out_vector
) noexcept {
    if (in_vector_0.size() != in_vector_1.size()) {
        _putws(L"Both input vectors must be os the same size!");
        return thrust::host_vector<T> {};
    }
    thrust::device_vector<T> dev_0 { in_vector_0.begin(), in_vector_0.end() };
    thrust::device_vector<T> dev_1 { in_vector_1.begin(), in_vector_1.end() };
    thrust::device_vector<T> dev_res { out_vector.begin(), out_vector.end() };

    SumKernel<T><<<1, in_vector_0.size()>>>(dev_0, dev_1, dev_res);
    hipDeviceSynchronize();
    return thrust::host_vector<T> { dev_res };
}

int main(void) {
    ::srand(time(nullptr));

    std::vector<int64_t> randoms_0(300);
    std::vector<int64_t> randoms_1(300);
    std::for_each(randoms_0.begin(), randoms_0.end(), [](int64_t& r_0) { r_0 = ::rand(); });
    std::for_each(randoms_1.begin(), randoms_1.end(), [](int64_t& r_1) { r_1 = ::rand(); });

    std::vector<int64_t> sums(300);
    for (std::size_t i {}; i < 300; ++i) sums.at(i) = randoms_0.at(i) + randoms_1.at(i);

    auto device_results { KernelManager<int64_t>(randoms_0, randoms_1, sums) };

    _putws(L"  Host sum       ||  Device sum      \n");
    for (std::size_t i {}; i < 300; ++i) wprintf_s(L"%10lld    %10lld\n", sums.at(i), device_results[i]);
    return EXIT_SUCCESS;
}